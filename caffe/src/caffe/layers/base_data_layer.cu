#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template<typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    // Reshape to loaded data.
    top[0]->ReshapeLike(batch->data_);
    // Copy the data
    caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
        top[0]->mutable_gpu_data());
    if (this->output_labels_) {
      // Reshape to loaded labels.
      top[1]->ReshapeLike(batch->label_);
      // Copy the labels.
      caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
          top[1]->mutable_gpu_data());
    }
    // Ensure the copy is synchronous wrt the host, so that the next batch isn't
    // copied in meanwhile.
    CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());

    // Reshape to loaded data.
    top[0]->ReshapeLike(batch->data_);
    // Copy the data
    greentea_copy<Dtype>(batch->data_.count(),
                         (cl_mem) (batch->data_.gpu_data()), 0,
                         (cl_mem) (top[0]->mutable_gpu_data()), 0, &ctx);
    if (this->output_labels_) {
      // Reshape to loaded labels.
      top[1]->ReshapeLike(batch->label_);
      // Copy the labels.
      greentea_copy<Dtype>(batch->label_.count(),
                           (cl_mem) (batch->label_.gpu_data()), 0,
                           (cl_mem) (top[1]->mutable_gpu_data()), 0, &ctx);
    }
#endif  // USE_GREENTEA
  }

  prefetch_free_.push(batch);
}

<<<<<<< HEAD

template <typename Dtype>
void BasePrefetchingDenseDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // First, join the thread
  JoinPrefetchThread();
  // Reshape to loaded data.
  top[0]->ReshapeLike(this->prefetch_data_);
  // Copy the data
  caffe_copy(prefetch_data_.count(), prefetch_data_.cpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(prefetch_label_);
    // Copy the labels.
    caffe_copy(prefetch_label_.count(), prefetch_label_.cpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Start a new prefetch thread
  CreatePrefetchThread();
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDenseDataLayer);
=======
>>>>>>> add_extract_patch_module
INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
